#include "hip/hip_runtime.h"
#include <limits>
#include "ros/console.h"
//#include "hip/hip_runtime_api.h"
#include "base_trajectory/cuda_utils.h"
#include "base_trajectory/obstacle_gradient_impl_cuda.h"

ObstacleGradientImplCuda::ObstacleGradientImplCuda(int xs, int ys, uint8_t threshold)
	: ObstacleGradientImpl(xs, ys, threshold)
{
	ns_ = 0;
	setSize(xs, ys);
	cudaSafeCall(hipStreamCreateWithFlags(&cudaStreamH_, hipStreamNonBlocking));
	cudaSafeCall(hipStreamCreateWithFlags(&cudaStreamV_, hipStreamNonBlocking));
	cudaSafeCall(hipStreamCreateWithFlags(&cudaStreamForGraph_, hipStreamNonBlocking));
	cudaSafeCall(hipEventCreate(&cudaEventInputReady_));
	cudaSafeCall(hipEventCreate(&cudaEventHKernel_));
	cudaSafeCall(hipEventCreate(&cudaEventVKernel_));
}

ObstacleGradientImplCuda::~ObstacleGradientImplCuda()
{
	freeDeviceMemory();
	cudaSafeCall(hipStreamDestroy(cudaStreamH_));
	cudaSafeCall(hipStreamDestroy(cudaStreamV_));
	cudaSafeCall(hipStreamDestroy(cudaStreamForGraph_));
	cudaSafeCall(hipEventDestroy(cudaEventInputReady_));
	cudaSafeCall(hipEventDestroy(cudaEventHKernel_));
	cudaSafeCall(hipEventDestroy(cudaEventVKernel_));
	//hipDeviceReset();
	//hipProfilerStop();
}

__global__
void calcHorizontal(const int nx, int ny, const unsigned char* costs, const uint8_t threshold, const unsigned int maxUInt, unsigned int* hPotentials)
{
	// Each thread handles an individual row
	const int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= ny)
		return;
	const int startIdx = row * nx;
	for (int i = startIdx; i < startIdx + nx; i++)
	{
		if (costs[i] <= threshold)
		{
			hPotentials[i] = 0;
		}
		else
		{
			hPotentials[i] = maxUInt;
		}
	}
	// For this row iteratively propagate potentials from left and right of each cell
	for (int i = startIdx + 1; i < (startIdx + nx); i++)
	{
		hPotentials[i] = min(hPotentials[i-1] + 1, hPotentials[i]);
	}
	// And then repeat moving right to left
	for (int i = startIdx + nx - 2; i >= startIdx; i--)
	{
		hPotentials[i] = min(hPotentials[i+1] + 1, hPotentials[i]);
	}
}

__global__
void calcVertical(const int nx, const int ny, const unsigned char* costs, const uint8_t threshold, const unsigned int maxUInt, unsigned int* vPotentials)
{
	// Each thread handles an individual col
	const int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (col >= nx)
		return;
	const int startIdx = col;
	for (int row = 0; row < ny; row++)
	{
		const auto i = startIdx + row * nx;
		if (costs[i] <= threshold)
		{
			vPotentials[i] = 0;
		}
		else
		{
			vPotentials[i] = maxUInt;
		}
	}

	for (int row = 1; row < ny; row++)
	{
		const auto i = startIdx + row * nx;
		vPotentials[i] = min(vPotentials[i-nx] + 1, vPotentials[i]);
	}
	for (int row = ny - 2; row >= 0; row--)
	{
		const auto i = startIdx + row * nx;
		vPotentials[i] = min(vPotentials[i+nx] + 1, vPotentials[i]);
	}
}

__global__
void sumHorizonalVertical(const int nx, const int ny, unsigned int* hPotentials, const unsigned int* vPotentials, const uint8_t *costs)
{
	// Each thread handles an individual row
	const int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= ny)
		return;

	const int startIdx = row * nx;
	for (int i = startIdx; i < startIdx + nx; i++)
	{
		hPotentials[i] += vPotentials[i];
		hPotentials[i] *= costs[i];
	}
}

// Given the grid map of size nx_, ny_ containing costs costs, create a potential
// map of the same space. Each grid potential is defined to be 0 if there is no
// obstacle at the grid location.  If there is an obstacle, the potential is the
// sum of the shortest horizontal and vertical distances to a grid location with
// no obstacle. This will tend to produce potential fields which look like pyramids,
// with a hopefully easy to follow gradient from obstacles into cells which are
// not obstructed.
void ObstacleGradientImplCuda::calculatePotentials(const unsigned char* costs, unsigned int* potentials)
{
	constexpr size_t THREADS_PER_BLOCK = 128;
	// memcpy h2d costs
	if (recordCudaGraph_)
	{
		cudaSafeCall(hipStreamBeginCapture(cudaStreamH_, hipStreamCaptureModeGlobal));
		cudaSafeCall(hipMemcpyAsync(deviceCosts_, costs, ns_ * sizeof(*costs), hipMemcpyHostToDevice, cudaStreamH_));
		cudaSafeCall(hipEventRecord(cudaEventInputReady_, cudaStreamH_));
		// calc horizontal
		calcHorizontal<<<numBlocks(THREADS_PER_BLOCK, ny_), THREADS_PER_BLOCK, 0, cudaStreamH_>>>(nx_, ny_, deviceCosts_, threshold_, std::numeric_limits<unsigned int>::max(), deviceHPotentials_);
		cudaSafeCall(hipEventRecord(cudaEventHKernel_, cudaStreamH_));
		// calc vertical
		cudaSafeCall(hipStreamWaitEvent(cudaStreamV_, cudaEventInputReady_, 0));
		calcVertical<<<numBlocks(THREADS_PER_BLOCK, nx_), THREADS_PER_BLOCK, 0, cudaStreamV_>>>(nx_, ny_, deviceCosts_, threshold_, std::numeric_limits<unsigned int>::max(), deviceVPotentials_);
		cudaSafeCall(hipEventRecord(cudaEventVKernel_, cudaStreamV_));

		// sum h+v into h once both the sub-sums are complete
		cudaSafeCall(hipStreamWaitEvent(cudaStreamH_, cudaEventHKernel_, 0)); // redundant?
		cudaSafeCall(hipStreamWaitEvent(cudaStreamH_, cudaEventVKernel_, 0));
		sumHorizonalVertical<<<numBlocks(THREADS_PER_BLOCK, ny_), THREADS_PER_BLOCK, 1, cudaStreamH_>>>(nx_, ny_, deviceHPotentials_, deviceVPotentials_, deviceCosts_);

		// memcpy async d2h device Hpotentials into host potentials
		cudaSafeCall(hipMemcpyAsync(potentials, deviceHPotentials_, ns_ * sizeof(*potentials), hipMemcpyDeviceToHost, cudaStreamH_));
		cudaSafeCall(hipStreamEndCapture(cudaStreamH_, &cudaGraph_));
		cudaSafeCall(hipGraphInstantiate(&cudaGraphExec_, cudaGraph_, 0, 0, 0));
		recordCudaGraph_ = false;
		ROS_INFO_STREAM("Graph created");
	}

	cudaSafeCall(hipGraphLaunch(cudaGraphExec_, cudaStreamForGraph_));
	cudaSafeCall(hipStreamSynchronize(cudaStreamForGraph_));
}

void ObstacleGradientImplCuda::setSize(int nx, int ny)
{
	const auto ns = ns_;
	ObstacleGradientImpl::setSize(nx, ny);

	if (ns != ns_)
	{
		freeDeviceMemory();
		cudaSafeCall(hipMalloc(&deviceHPotentials_, ns_ * sizeof(*deviceHPotentials_)));
		cudaSafeCall(hipMalloc(&deviceVPotentials_, ns_ * sizeof(*deviceVPotentials_)));
		cudaSafeCall(hipMalloc(&deviceCosts_, ns_ * sizeof(*deviceCosts_)));
		recordCudaGraph_ = true;
	}
}

void ObstacleGradientImplCuda::freeDeviceMemory()
{
	if (deviceHPotentials_)
	{
		cudaSafeCall(hipFree(deviceHPotentials_));
		deviceHPotentials_ = nullptr;
	}
	if (deviceVPotentials_)
	{
		cudaSafeCall(hipFree(deviceVPotentials_));
		deviceVPotentials_ = nullptr;
	}
	if (deviceCosts_)
	{
		cudaSafeCall(hipFree(deviceCosts_));
		deviceCosts_ = nullptr;
	}
}
